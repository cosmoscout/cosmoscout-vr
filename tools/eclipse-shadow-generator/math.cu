#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
//                               This file is part of CosmoScout VR                               //
////////////////////////////////////////////////////////////////////////////////////////////////////

// SPDX-FileCopyrightText: German Aerospace Center (DLR) <cosmoscout@dlr.de>
// SPDX-License-Identifier: MIT

#include "math.cuh"

namespace math {

////////////////////////////////////////////////////////////////////////////////////////////////////

double __host__ __device__ angleBetweenVectors(glm::dvec3 const& u, glm::dvec3 const& v) {
  return 2.0 * glm::asin(0.5 * glm::length(u - v));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

glm::dvec3 __host__ __device__ rotateVector(
    glm::dvec3 const& v, glm::dvec3 const& a, double cosMu) {
  double sinMu = glm::sqrt(1.0 - cosMu * cosMu);
  return v * cosMu + glm::cross(a, v) * sinMu + a * glm::dot(a, v) * (1.0 - cosMu);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

double __host__ __device__ getCircleArea(double r) {
  return glm::pi<double>() * r * r;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

double __host__ __device__ getCapArea(double r) {
  return 2.0 * glm::pi<double>() * (1.0 - std::cos(r));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

double __host__ __device__ getCapIntersection(double rSun, double rOcc, double d) {
  d = std::abs(d);

  if (rSun <= 0.0 || rOcc <= 0.0) {
    return 0.0;
  }

  if (d >= rSun + rOcc) {
    return 0.0;
  }

  if (d <= std::abs(rOcc - rSun)) {
    return getCapArea(glm::min(rSun, rOcc));
  }

  // clang-format off
  return 2.0 * (glm::pi<double>() -
      std::acos((std::cos(d)    - std::cos(rSun) * std::cos(rOcc)) / (std::sin(rSun) * std::sin(rOcc)))
    - std::acos((std::cos(rOcc) - std::cos(d)    * std::cos(rSun)) / (std::sin(d)    * std::sin(rSun))) * std::cos(rSun)
    - std::acos((std::cos(rSun) - std::cos(d)    * std::cos(rOcc)) / (std::sin(d)    * std::sin(rOcc))) * std::cos(rOcc));
  // clang-format on
}

////////////////////////////////////////////////////////////////////////////////////////////////////

double __host__ __device__ getCircleIntersection(double rSun, double rOcc, double d) {
  d = std::abs(d);

  if (rSun <= 0.0 || rOcc <= 0.0) {
    return 0.0;
  }

  if (d >= rSun + rOcc) {
    return 0.0;
  }

  if (d <= std::abs(rOcc - rSun)) {
    return getCircleArea(glm::min(rSun, rOcc));
  }

  double d1 = (rSun * rSun - rOcc * rOcc + d * d) / (2 * d);
  double d2 = d - d1;

  return rSun * rSun * std::acos(d1 / rSun) - d1 * std::sqrt(rSun * rSun - d1 * d1) +
         rOcc * rOcc * std::acos(d2 / rOcc) - d2 * std::sqrt(rOcc * rOcc - d2 * d2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

double __host__ __device__ sampleCircleIntersection(
    double rSun, double rOcc, double d, common::LimbDarkening const& limbDarkening) {

  // Sanity checks.
  d = std::abs(d);
  if (rSun <= 0.0 || rOcc <= 0.0) {
    return 0.0;
  }

  // There is no overlapping at all.
  if (d >= rSun + rOcc) {
    return 0.0;
  }

  // The Sun is fully occluded.
  if (d + rSun <= rOcc) {
    return getCircleArea(rSun);
  }

  // We sample a rectangular region which covers the upper half of the intersection area.
  double sampleAreaMinX = d - rOcc;
  double sampleAreaMaxX = glm::min(rSun, d + rOcc);
  double sampleAreaMinY = 0.0;
  double sampleAreaMaxY = glm::min(rSun, rOcc);

  // If both circles are so much apart, we do not have to sample up to
  // glm::min(rSun, rOcc) vertically. We get the required sample height with Heron's
  // formula.
  if (d * d + std::pow(glm::min(rSun, rOcc), 2.0) > std::pow(glm::max(rSun, rOcc), 2.0)) {
    double a       = rSun;
    double b       = rOcc;
    double c       = d;
    double s       = 0.5 * (a + b + c);
    sampleAreaMaxY = 2.0 * std::sqrt(s * (s - a) * (s - b) * (s - c)) / c;
  }

  const int32_t xSamples = 512;
  const int32_t ySamples = xSamples / 2;
  double        area     = 0.0;

  glm::dvec2 samplePos;

  for (int32_t y(0); y < ySamples; ++y) {
    samplePos.y = (1.0 * y + 0.5) / ySamples;
    samplePos.y = sampleAreaMinY + samplePos.y * (sampleAreaMaxY - sampleAreaMinY);

    for (int32_t x(0); x < xSamples; ++x) {
      samplePos.x = (1.0 * x + 0.5) / xSamples;
      samplePos.x = sampleAreaMinX + samplePos.x * (sampleAreaMaxX - sampleAreaMinX);

      if (glm::distance2(glm::dvec2(0.0, 0.0), samplePos) < rSun * rSun &&
          glm::distance2(glm::dvec2(d, 0.0), samplePos) < rOcc * rOcc) {
        area += (sampleAreaMaxX - sampleAreaMinX) / xSamples * (sampleAreaMaxY - sampleAreaMinY) /
                ySamples * limbDarkening.get(glm::length(samplePos) / rSun);
      }
    }
  }

  // We sampled only half the area.
  return area * 2.0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void __host__ __device__ mapPixelToRadii(glm::ivec2 const& pixel, uint32_t resolution,
    common::Mapping const& mapping, double& radiusOcc, double& distance) {

  double x = glm::pow((1.0 * pixel.x + 0.5) / resolution, mapping.mExponent);
  double y = 1.0 - glm::pow(1.0 - (1.0 * pixel.y + 0.5) / resolution, mapping.mExponent);

  double radiusSun = 1.0;
  radiusOcc        = radiusSun / x - radiusSun;

  double minDistance = mapping.mIncludeUmbra ? 0.0 : glm::max(radiusOcc - radiusSun, 0.0);
  double maxDistance = radiusOcc + radiusSun;

  distance = minDistance + y * (maxDistance - minDistance);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

uint32_t __host__ __device__ mapPixelToAngles(glm::ivec2 const& pixel, uint32_t resolution,
    common::Mapping const& mapping, common::Geometry const& geometry, double& phiOcc,
    double& phiSun, double& delta) {

  // This methods computes two circles, one representing the Sun and the other the occluder, as well
  // as the distance between their centers. All values are scaled in such a way/ that the radius of
  // the Solar disc is 1.0.
  double radiusOcc, distance;
  mapPixelToRadii(pixel, resolution, mapping, radiusOcc, distance);

  // To compute the actual geometry of the involved bodies, we need to find the position in space
  // where the anguala radii and the anualar distance between the Sun and the occluder are as
  // computed above, scaled by an unknown factor.

  // As an initial guess, we assume that the Sun appears as large as it does from the occluder's
  // position. So we scale all values to this.
  phiSun = glm::asin(geometry.mRadiusSun / geometry.mSunOccDist);
  phiOcc = glm::min(glm::pi<double>() / 2.0, radiusOcc * phiSun);
  delta  = glm::min(glm::pi<double>() / 2.0, distance * phiSun);

  // If the occluder is larger than pi/2, an impossible situation is given.
  if (phiOcc >= glm::pi<double>() / 2.0) {
    return 0;
  }

  double   error      = 1.0;
  uint32_t iterations = 0;

  while (error > 0.0001 && ++iterations < 100) {

    // Compute how far we would be from the occluder if it appeared this large.
    double occDist = geometry.mRadiusOcc / glm::sin(phiOcc);

    // Given the angular distance between the Sun and the occluder, we can compute the distance
    // between the Sun and the searched point. This will be farther away than the initial guess.
    double sunDist = occDist * glm::cos(delta) +
                     glm::sqrt(occDist * occDist * glm::cos(delta) * glm::cos(delta) -
                               occDist * occDist + geometry.mSunOccDist * geometry.mSunOccDist);

    // Using the real radius of the Sun, we can now compute again how large the Sun will appear from
    // the searched point. We use this as a new guess.
    double newPhiSun = glm::asin(geometry.mRadiusSun / sunDist);
    double newPhiOcc = radiusOcc * newPhiSun;
    double newDelta  = distance * newPhiSun;

    // Compute the maximum error in all three values.
    error = glm::max(glm::abs(phiSun - newPhiSun) / phiSun,
        glm::max(glm::abs(phiOcc - newPhiOcc) / phiOcc, glm::abs(delta - newDelta) / delta));

    phiSun = newPhiSun;
    phiOcc = newPhiOcc;
    delta  = newDelta;
  }

  return iterations;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace math
